#include "hip/hip_runtime.h"
#include <iostream>
#include <iomanip>
#include <cstdlib>
#include <chrono>
#include <cstdlib>
#include <algorithm>
//INSIRA AS IMPORTACOES NECESSARIAS DA THRUST
#include <cmath>
#include <random>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/generate.h>
#include <thrust/functional.h>
#include <thrust/copy.h>

using namespace std::chrono;

void reportTime(const char* msg, steady_clock::duration span) {
    auto ms = duration_cast<milliseconds>(span);
    std::cout << msg << " - levou - " <<
    ms.count() << " milisegundos" << std::endl;
}

// CRIE UMA FUNCTOR PARA CALCULAR A SQUARE
struct own_square {
    __host__ __device__
    float operator()(const float &x) const {
        return x * x;
    }
};

// IMPLEMENTE O CALCULO DA MAGNITUDE COM THRUST
float magnitude(thrust::device_vector<float>& v) {
    float result;
    thrust::transform(v.begin(), v.end(), v.begin(), own_square());
    result = thrust::reduce(v.begin(), v.end());
    result = sqrt(result);
    return result;
    // alternativa usando fusion
    // std::sqrt(thrust::transform_reduce( v.begin(), v.end(), square(), 0.0, plus()));
}

int main(int argc, char** argv) {
    if (argc != 2) {
        std::cerr << argv[0] << ": numero invalido de argumentos\n"; 
        std::cerr << "uso: " << argv[0] << "  tamanho do vetor\n"; 
        return 1;
    }
    int n = std::atoi(argv[1]); //numero de elementos
    steady_clock::time_point ts, te;

    // Faça um  vector em thrust 
    thrust::device_vector<float> v_d(n);

    // inicilize o  vector
    ts = steady_clock::now();

    // Preencha o vetor com valores aleatorios
    thrust::generate(v_d.begin(), v_d.end(), rand);

    te = steady_clock::now();
    reportTime("Inicializacao", te - ts);

    // Calcule a magnitude do vetor
    ts = steady_clock::now();
    float len = magnitude(v_d);
    te = steady_clock::now();
    reportTime("Tempo para calculo", te - ts);


    std::cout << std::fixed << std::setprecision(4);
    std::cout << "Magnitude : " << len << std::endl;
}